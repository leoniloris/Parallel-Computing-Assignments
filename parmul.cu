#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
# include <omp.h>
//#define TILE_WIDTH 10
//#define N_LOOP 1
#define MAX_MATRIX_SIZE 1024
int MATRIX_SIZE,TILE_WIDTH,N_LOOP;

__host__ double compare(double * A,double * B,int MATRIX_SIZE){
double err=0;
double er=0;
	for (int i=0;i<MATRIX_SIZE*MATRIX_SIZE;i++)
		{  er = (A[i]-B[i])*(A[i]-B[i]);
		//printf("%f\n",er);
			if(er<0.1);
			else err=+er;}
		return err;
}
// parallel multiplication aqui!!!!!
__global__ void matrixMultiplyPar(double * A, double * B, double * C,
  		       int numARows, int numAColumns,
			       int numBRows, int numBColumns,
			       int numCRows, int numCColumns,int TILE_WIDTH) {
	 //for(int a=0;a<numBColumns*numBColumns;a++) C[a]=0.0;
    extern __shared__ double ds_MN[];
    int bx = blockIdx.x, by = blockIdx.y,
       tx = threadIdx.x, ty = threadIdx.y,
       Row = by * TILE_WIDTH + ty,
       Col = bx * TILE_WIDTH + tx;
    double Pvalue = 0.0;
    for (int m = 0; m < (numAColumns-1)/TILE_WIDTH+1; ++m) {
       if (Row < numARows && m*TILE_WIDTH+tx < numAColumns) ds_MN[ty*TILE_WIDTH+tx]= A[Row*numAColumns + m*TILE_WIDTH+tx];  //ds_M[ty*TILE_WIDTH+tx] = A[Row*numAColumns + m*TILE_WIDTH+tx];
       else  ds_MN[ty*TILE_WIDTH+tx] = 0.0;
       if (Col < numBColumns && m*TILE_WIDTH+ty < numBRows) ds_MN[TILE_WIDTH*TILE_WIDTH+ty*TILE_WIDTH+tx] = B[(m*TILE_WIDTH+ty)*numBColumns+Col];//ds_N[ty*TILE_WIDTH+tx] = B[(m*TILE_WIDTH+ty)*numBColumns+Col];
       else ds_MN[TILE_WIDTH*TILE_WIDTH+ty*TILE_WIDTH+tx]= 0.0;
       __syncthreads();
       for (int k = 0; k < TILE_WIDTH; ++k)
          Pvalue += ds_MN[ty*TILE_WIDTH+k] * ds_MN[TILE_WIDTH*TILE_WIDTH+k*TILE_WIDTH+tx];
       __syncthreads();
    }
    if (Row < numCRows && Col < numCColumns)
       C[Row*numCColumns+Col] = Pvalue;
       
}
// blocked sequencial multiplication aqui!!!!! (tileado)
__host__ void matrixMultiply(double * A, double * B, double * C,int numARows, int numAColumns,int numBRows, int numBColumns){
	for(int a=0;a<numBColumns*numBColumns;a++) C[a]=0.0;
	for ( int i=0; i<numAColumns; i+=TILE_WIDTH )
        for (int  j=0; j<numAColumns; j+=TILE_WIDTH )
            for (int k=0; k<numAColumns; k+=TILE_WIDTH )
                for (int y=i; y<min(i+TILE_WIDTH,numAColumns); y++ )
                    for (int x=j; x<min(j+TILE_WIDTH,numAColumns); x++ )
                        for (int z=k; z<min(k+TILE_WIDTH,numAColumns); z++ )
                            C[y*numAColumns+x] += A[y*numAColumns+z]*B[z*numAColumns+x];
                            
                            
                            
}
void Rand_matrix(double * M, int rows,int cols){	
	srand(time(NULL));
#pragma omp parallel for 
	for(int i=0;i<rows*cols;i++)
		M[i]=((double)(rand()%101)/100.00) - 0.5;
}
int main(int argc, char ** argv) {
	FILE *fp; // write results in a file
	if(argc<4){ // caso nao venha com argumentos
		printf(" Argumentos padrao selecionados\n resultados em 'results.dat'\n TILE_WIDTH=16\n cada multiplicaçao é executada apenas 2 vezes \n"); 
		fp = fopen("results.dat", "w+");
		TILE_WIDTH=16;
		N_LOOP=2;
	}
	else {
	fp = fopen(argv[1], "w+");
	TILE_WIDTH=atoi(argv[2]);
	N_LOOP=atoi(argv[3]);
	}
	hipEvent_t start, stop;// so I can save performance information
	
	double *hostA, *hostB, *hostC,*deviceA,*deviceB,*deviceC,*hostC_seq;


	hipEventCreate(&start);
	hipEventCreate(&stop);
	int numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns;	
	
	
	for (MATRIX_SIZE=8;MATRIX_SIZE<=MAX_MATRIX_SIZE ;MATRIX_SIZE*=2){
		float ms=0;
		numARows=numAColumns=numBRows=numBColumns=numCRows=numCColumns=MATRIX_SIZE;
		hostB=(double *)malloc(sizeof(double)*numBRows*numBColumns);
		hostA=(double *)malloc(sizeof(double)*numARows*numAColumns);
		hostC = (double *)malloc(sizeof(double) * numARows * numBColumns);
		hostC_seq = (double *)malloc(sizeof(double) * numARows * numBColumns);
		Rand_matrix(hostA, numARows, numAColumns);
		Rand_matrix(hostB, numARows, numAColumns);
		hipMalloc(&deviceA, sizeof(double) * numARows * numAColumns);
		hipMalloc(&deviceB, sizeof(double) * numBRows * numBColumns);
		hipMalloc(&deviceC, sizeof(double) * numCRows * numCColumns);
		hipMemcpy(deviceA, hostA, sizeof(double) * numARows * numAColumns, hipMemcpyHostToDevice);
		hipMemcpy(deviceB, hostB, sizeof(double) * numBRows * numBColumns, hipMemcpyHostToDevice);
		dim3 dimGrid((numCColumns-1)/TILE_WIDTH+1, (numCRows-1)/TILE_WIDTH+1, 1);
		dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

		
		// esta parte eu s`o vou executar qdo eu encontrar os valores `otimos
		hipEventRecord(start);
		#pragma omp parallel for firstprivate (hostA,hostB,hostC_seq)
		for(int vez=1;vez<=N_LOOP;vez++){
		#pragma omp critical
			matrixMultiply(hostA,hostB,hostC_seq,numARows, numAColumns, numBRows, numBColumns);
//for(int az=0;az<numCRows*numCRows;az++) printf("%f\n",hostC_seq[az]);
			}
		#pragma omp barrier	
		hipEventRecord(stop);hipEventSynchronize(stop);hipEventElapsedTime(&ms, start, stop);
		fprintf(fp,"%f ",ms/1000.0);
		
		/* @@@@@@@@@@@@@@@@@  parallel multiplication @@@@@@@@@@@@@@@@@ */
		hipEventRecord(start);
		#pragma omp parallel for firstprivate ( deviceA,deviceB,deviceC)
		for(int vez=1;vez<=N_LOOP;vez++){
		#pragma omp critical
			matrixMultiplyPar<<<dimGrid, dimBlock,2*TILE_WIDTH*TILE_WIDTH*sizeof(double)>>>( deviceA,deviceB,deviceC,numARows,numAColumns,numBRows,numBColumns,numCRows,numCColumns,TILE_WIDTH);
			hipDeviceSynchronize();
		}
		#pragma omp barrier
		hipEventRecord(stop);hipEventSynchronize(stop);hipEventElapsedTime(&ms, start, stop);
		hipMemcpy(hostC, deviceC, sizeof(double) * numCRows * numCColumns, hipMemcpyDeviceToHost);
		//for(int az=0;az<numCRows*numCRows;az++) printf("%f\n",hostC[az]);
		fprintf(fp,"%f\n",ms/1000.0);
		
		printf("erro acumulado %f\n",compare(hostC_seq,hostC,MATRIX_SIZE));
		
//for(int az=0;az<numCRows*numCRows;az++) printf("%f %f\n",hostC_seq[az],hostC[az]);
		free(hostA);
		free(hostB);
		free(hostC);
		free(hostC_seq);
		hipFree(deviceA);	
		hipFree(deviceB);	
		hipFree(deviceC);	
		printf("%d out of %d \n",MATRIX_SIZE,MAX_MATRIX_SIZE );
	}	
	

	fclose(fp);
	return 0;	



}

